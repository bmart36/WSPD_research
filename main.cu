#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <fstream>
#include <sstream>
#include "rounding.cu"
#include "shuffle.cu"
#include "build_tree.cu"
#include "wspd.cu"

using namespace std;

void readFile(const char *filename, float *points)
{
	string s, t, tmp;
	int n = 0;
	std::ifstream fin(filename);
	if (!fin.is_open()) cout << "Unable to open file!\n";

	while ( getline(fin, s) )
	{
		stringstream myline(s);
		myline >> t;
		if (t == "v")
		{
	        myline >> tmp;
			float x=stof(tmp.substr(0, tmp.find("/")));

			myline >> tmp;
			float y=stof(tmp.substr(0, tmp.find("/")));

			myline >> tmp;
			float z=stof(tmp.substr(0, tmp.find("/")));
			points[n] = x;
			points[n+1] = y;
			points[n+2] = z;
	        n+=3;
	 	}
	}
}

int main()
{
	unsigned int n = 12;

	// host variables
	float *h_f_points;
	long int *h_s_points;
	Node *h_root;
	Stack<Node> *h_quadtree = new Stack<Node>();
	Stack<Node2> *h_pairs = new Stack<Node2>();

	// device variables
	float *d_f_points;
	int *d_points;
	long int *d_s_points;
	Node *d_root;
	Stack<Node> *d_quadtree;
	Stack<Node> *d_roots;
	Stack<Node2> *d_queue;
	Stack<Node2> *wspd_pairs;

	// allocate host memory
	h_f_points = (float *)malloc(n * 3 * sizeof(float));
	h_s_points = (long int *)malloc(n * sizeof(long int));
	h_root = (Node *)malloc(sizeof(Node));

	// allocate device memory
	hipMalloc((void **) &d_f_points, n * 3 * sizeof(float));
	hipMalloc((void **) &d_points, n * 3 * sizeof(int));
	hipMalloc((void **) &d_s_points, n * sizeof(long int));
	hipMalloc((void **) &d_root, sizeof(Node));
	hipMalloc((void **) &d_quadtree, sizeof(Stack<Node>));
	hipMalloc((void **) &d_roots, sizeof(Stack<Node>));
	hipMalloc((void **) &d_queue, sizeof(Stack<Node2>));
	hipMalloc((void **) &wspd_pairs, sizeof(Stack<Node2>));

	// fill h_f_points array
	readFile("test.txt", h_f_points);

	// copy varibles to GPU
	hipMemcpy(d_roots, h_quadtree, sizeof(Stack<Node>), hipMemcpyHostToDevice);
	hipMemcpy(d_quadtree, h_quadtree, sizeof(Stack<Node>), hipMemcpyHostToDevice);
	hipMemcpy(d_f_points, h_f_points, n * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_queue, h_pairs, sizeof(Stack<Node2>), hipMemcpyHostToDevice);
	hipMemcpy(wspd_pairs, h_pairs, sizeof(Stack<Node2>), hipMemcpyHostToDevice);

	// rounding points to a grid
	rounding<<<1, n>>>(d_f_points, d_points, d_roots);

	// shuffle points' coordinates
	shuffle<<<1, n>>>(d_points, d_s_points);

	// sort the points in invreasing order
	thrust::device_ptr<long int> t_d_s(d_s_points);
	if(!thrust::is_sorted(t_d_s, t_d_s + (n-1)))
	{
		thrust::stable_sort(t_d_s, t_d_s + n);
	}
 
	// copy ordered points to CPU
	hipMemcpy(h_s_points, d_s_points, n * sizeof(long int), hipMemcpyDeviceToHost);

	// size of the grid
	int w = range(h_s_points[n-1]);

	// computing compressed quadtrees
	compressed_quadtree<<< 1, BLOCK_SIZE>>>(d_s_points, n/BLOCK_SIZE, d_roots, d_quadtree, w, msb(h_s_points[n-1]));

	// merging of compressed quadtrees
	tree_merge<<<1 ,1>>>(d_roots, d_quadtree, d_root, w, msb(h_s_points[n-1]));

	// copy quadtree to CPU
	hipMemcpy(h_quadtree, d_quadtree, sizeof(Stack<Node>), hipMemcpyDeviceToHost);
	//hipMemcpy(*h_root, d_root, sizeof(Node), hipMemcpyDeviceToHost);


	// computing WSPD
	pre_wspd<<<1, BLOCK_SIZE>>>(d_quadtree, d_queue);
	wspd<<<1, 1>>>(1, d_queue, wspd_pairs);

	hipFree(d_f_points);
	hipFree(d_points);
	hipFree(d_s_points);
	hipFree(d_roots);
	hipFree(d_quadtree);
	hipFree(d_queue);
	hipFree(wspd_pairs);

	// return 0;
}

